#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <time.h>

        __global__
void kern_RGBtoRGBA(unsigned char *dst,
                size_t dstPitch,
                unsigned char *src,
                size_t srcPitch,
                size_t width,
                size_t height)
{
        const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if(x >= width)
                return;

        if(y >= height)
                return;

        uchar3 *src_px = (uchar3 *) (src + y * srcPitch) + x;
        uchar4 *dst_px = (uchar4 *) (dst + y * dstPitch) + x;

        const uchar3 px = *src_px;

        *dst_px = make_uchar4(px.x, px.y, px.z, 0);
}

#define max(a, b)      (((a) > (b))? (a): (b))
#define min(a, b)      (((a) < (b))? (a): (b))

__global__
void kern_UYVYtoRGBA(unsigned char *dst,
		size_t dstPitch,
		unsigned char *src,
		size_t srcPitch,
		size_t width,
		size_t height)
{
	const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if(x >= width)
		return;

	if(y >= height)
		return;

	const int uyvy_block = x / 2;

	uchar4 *src_px = (uchar4 *) (src + y * srcPitch) + uyvy_block;
	uchar4 *dst_px = (uchar4 *) (dst + y * dstPitch) + x;

	const uchar4 block = *src_px;

	int u = block.x;
	int v = block.z;
	int luma = (x % 2) ? block.w : block.y;

	int r = min(max(1.164f*(luma - 16) + 1.793f*(v - 128), 0), 255);
	int g = min(max(1.164f*(luma - 16) - 0.534f*(v - 128) - 0.213f*(u - 128), 0), 255); 
	int b = min(max(1.164f*(luma - 16) + 2.115f*(u - 128), 0), 255);

	*dst_px = make_uchar4(r, g, b, 0);
}

void cuda_RGBto_RGBA(unsigned char *dst,
                size_t dstPitch,
                unsigned char *src,
                size_t srcPitch,
                size_t width,
                size_t height,
                ihipStream_t *stream){

        dim3 blockSize(32,32);
        dim3 numBlocks((width + blockSize.x - 1) / blockSize.x,
                        (height + blockSize.y - 1) / blockSize.y);

        kern_RGBtoRGBA<<<numBlocks, blockSize, 0, stream>>>(dst, dstPitch, src, srcPitch, width, height);
}

void cuda_UYVY_to_RGBA(unsigned char *dst,
                size_t dstPitch,
                unsigned char *src,
                size_t srcPitch,
                size_t width,
                size_t height,
                ihipStream_t *stream){

        dim3 blockSize(32,32);
        dim3 numBlocks((width + blockSize.x - 1) / blockSize.x,
                        (height + blockSize.y - 1) / blockSize.y);

        kern_UYVYtoRGBA<<<numBlocks, blockSize, 0, stream>>>(dst, dstPitch, src, srcPitch, width, height);
}
